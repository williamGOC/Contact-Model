#include <stdio.h> 
#include <stdlib.h> 
#include <string.h> 

#include <GL/gl.h> 
#include <GL/glut.h> 
#include <cuda_gl_interop.h> 
#include <GL/glext.h> 
#include <GL/glx.h> 

#define REFRESH_DELAY     10 //ms

#define GET_PROC_ADDRESS( str ) glXGetProcAddress( (const GLubyte *)str ) 

static void HandleError( hipError_t err, const char *file,  int line ) { 
    if (err != hipSuccess) { 
            printf( "%s in %s at line %d\n", hipGetErrorString( err ),  file, line ); 
            exit( EXIT_FAILURE ); 
    } 
} 
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ )) 


PFNGLBINDBUFFERARBPROC    glBindBuffer     = NULL; 
PFNGLDELETEBUFFERSARBPROC glDeleteBuffers  = NULL; 
PFNGLGENBUFFERSARBPROC    glGenBuffers     = NULL; 
PFNGLBUFFERDATAARBPROC    glBufferData     = NULL; 


GLuint  bufferObj; 
hipGraphicsResource *resource; 

// define la interface entre opengl y mi sistema //////
#include "CP_GPU.h"
#include "interfaz.h"

static void draw_func( void ) { 
  	
  	rescale();
  	change_pixels();
	
	glDrawPixels(DIM, DIM, GL_RGBA, GL_UNSIGNED_BYTE, 0); 
	glutSwapBuffers(); 
}

static void key_func( unsigned char key, int r, int l) { 
  switch (key) { 
  	case 27: 
        HANDLE_ERROR( hipGraphicsUnregisterResource( resource ) ); 
        glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, 0 ); 
        glDeleteBuffers( 1, &bufferObj ); 
        exit(0); 
        break;
    case 'T':
        if(x - XINC > 0) x -= XINC; 
	Sptr->SetX(x);        
	std::cout << "x = " << x << std::endl;
        break;    
    case 't':
        x+=XINC; 
	Sptr->SetX(x);        
	std::cout << "x = " << x << std::endl;
        break;    
    default:
        break;
  } 
} 


////////////////////////////////////////////////////////////////////////////////
//! Mouse event handlers
////////////////////////////////////////////////////////////////////////////////
// mouse controls
int mouse_old_x, mouse_old_y;
int mouse_buttons = 0;

float rotate_x = 0.0, rotate_y = 0.0;
float translate_z = -3.0;

void mouse(int button, int state, int x, int y)
{
    if (state == GLUT_DOWN)
    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)
    {
        mouse_buttons = 0;
    }

    mouse_old_x = x;
    mouse_old_y = y;

    sqx=(x>0 && x<DIM)?(x):(sqx); sqy=(y>0 && y<DIM)?(y):(sqy);
    std::cout << sqx << " (clicks) " << sqy << " " << signo << std::endl;   
}



void timerEvent(int value)
{
    glutPostRedisplay();
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);
}



int main(int argc, char **argv) { 

	int args = 1;
	// size of the system (default 128)
	int L = (argc > args) ? (atoi(argv[args])):(512);
	if (L%2 == 1 || L < 0) {
		std::cout << "ERROR: L must be even and positive" << std::endl;
		exit(1);
	}
	args++;

	// temperature (default 2.0)
	float x = (argc > args) ? (atof(argv[args])):(1.5);
	args++;

	
	// iterations (default 1000)
	int trun = (argc > args) ? (atoi(argv[args])):(1000);
	args++;

	// global seed (default 0)
	int seed = (argc > args) ? (atoi(argv[args])):(0);
	srand(seed);

	std::cout << "L = " << L << std::endl;
	std::cout << "x = " << x << std::endl;
	std::cout << "trun = " << trun << std::endl;
	std::cout << "seed = " << seed << std::endl;

  ////// declarar/inicializar sistema //////
  ContactProcess S(L, x, seed);

  Sptr=&S;
  inicializar_variables_globales_sistema();
  //////////////////////////////////////////

  cudaGLSetGLDevice( 0 ); 

  glutInit( &argc, argv ); 
  glutInitDisplayMode( GLUT_DOUBLE | GLUT_RGBA ); 
  glutInitWindowSize( L, L ); 
  glutCreateWindow( "OpenGL: Contact Processes app" ); 

  glBindBuffer    = (PFNGLBINDBUFFERARBPROC)GET_PROC_ADDRESS("glBindBuffer"); 
  glDeleteBuffers = (PFNGLDELETEBUFFERSARBPROC)GET_PROC_ADDRESS("glDeleteBuffers"); 
  glGenBuffers    = (PFNGLGENBUFFERSARBPROC)GET_PROC_ADDRESS("glGenBuffers"); 
  glBufferData    = (PFNGLBUFFERDATAARBPROC)GET_PROC_ADDRESS("glBufferData"); 

  glGenBuffers( 1, &bufferObj ); 
  glBindBuffer( GL_PIXEL_UNPACK_BUFFER_ARB, bufferObj ); 
  glBufferData( GL_PIXEL_UNPACK_BUFFER_ARB, DIM * DIM * 4, NULL, GL_DYNAMIC_DRAW_ARB ); 


  hipGraphicsGLRegisterBuffer( &resource, bufferObj, cudaGraphicsMapFlagsNone ); 

// set up GLUT and kick off main loop 
 glutKeyboardFunc( key_func ); 
 glutDisplayFunc( draw_func ); 
 glutMouseFunc(mouse);

  glutTimerFunc(REFRESH_DELAY, timerEvent,0);
  glutMainLoop(); 
}
